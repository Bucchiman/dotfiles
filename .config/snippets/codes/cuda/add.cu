#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void addMatrix (array) {
  printf("%d\n", threadIdx.x);
}


int main(void) {
  float* array;
  hipMalloc((void**)&array, sizeof(float)*512*512);
  dim3 Dg(512, 1, 1);
  dim3 Db(512, 1, 1);
  addMatrix <<<Dg, Db>>>(array);
  hipDeviceSynchronize();
  hipMemcpy(host_matrix, dev_matrix, hipMemcpyDeviceToHost);

  hipFree((void*)array);
  return 0;
}
