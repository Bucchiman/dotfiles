
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<sys/time.h>
#define ARRAY_SIZE 512*512

__global__ void sumArraysOnGPU(float* A, float* B, float* C) {
  unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int idx = iy * 512 + ix;
  C[idx] = A[idx] + B[idx];
}


int main(void) {
  /* hostメモリ宣言 */
  float* h_A;
  float* h_B;
  float* h_C;

  /* hostメモリ確保 */
  h_A = (float*)malloc(ARRAY_SIZE*sizeof(float));
  h_B = (float*)malloc(ARRAY_SIZE*sizeof(float));
  h_C = (float*)malloc(ARRAY_SIZE*sizeof(float));

  /* deviceメモリの宣言 */
  float* d_A;
  float* d_B;
  float* d_C;

  /* deviceメモリの確保 */
  hipMalloc((float**)&d_A, ARRAY_SIZE*sizeof(float));
  hipMalloc((float**)&d_B, ARRAY_SIZE*sizeof(float));
  hipMalloc((float**)&d_C, ARRAY_SIZE*sizeof(float));

  /* hostメモリからdeviceメモリにデータ転送 */
  hipMemcpy(d_A, h_A, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_C, h_C, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);

  /* スレッド数の宣言 */
  dim3 block(512);
  dim3 grid(512);

  /* set the time */
  struct timeval tv, pe;
  gettimeofday(&tv, NULL);

  /* カーネル関数の呼び出し*/
  sumArraysOnGPU <<<block, grid>>>(d_A, d_B, d_C);

  /* 同期処理 */
  hipDeviceSynchronize();

  /* stop the timer */
  gettimeofday(&pe, NULL);
  float timer = pe.tv_sec - tv.tv_sec + (pe.tv_usec - tv.tv_usec)*1e-6;
  printf("%f sec\n", timer);

  /* deviceメモリからhostメモリにデータ転送 */
  hipMemcpy(h_C, d_C, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost);

  /* ホストメモリの解放 */
  free(h_A);
  free(h_B);
  free(h_C);

  /* デバイスメモリの解放 */
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  return 0;
}
