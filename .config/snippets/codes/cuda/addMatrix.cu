
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<sys/time.h>
#define ARRAY_SIZE_X 512
#define ARRAY_SIZE_Y 512

__global__ void sumArraysOnGPU(float* A, float* B, float* C){
  /* ここに､各スレッドの処理内容を記述*/
  unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int idx = iy * ARRAY_SIZE_X + ix;
  C[idx] = A[idx] + B[idx];
}

void initializeData(float *ip, int size){
  int i;
  for(i = 0; i < size; i++){
    ip[i] = 1.0;
  }
}

void checkResult(float *ip, int size){
  int i;
  int check = 0;
  
  for(i = 0; i < size; i ++){
    if(ip[i] != 2.0){
      check = 1;
    }
  }
  
  if(check == 0){
    printf("calc success!\n");
  }else{
    printf("calc result is not correct...\n");
  }
}

int main(void){
  
    
  /* hostメモリの宣言 */
  float *h_A, *h_B, *h_C;

  /* hostメモリの確保 */
  int total_array_size = ARRAY_SIZE_X * ARRAY_SIZE_Y;
  size_t nBytes = total_array_size * sizeof(float);
  
  h_A = (float *)malloc(nBytes);
  h_B = (float *)malloc(nBytes);
  h_C = (float *)malloc(nBytes);

  
  
  /* h_A  と h_Bを初期化 */
  initializeData(h_A, total_array_size);
  initializeData(h_B, total_array_size);
  

  /* deviceメモリの宣言　*/
  float *d_A, *d_B, *d_C;

  /* deviceメモリの確保 */  
  hipMalloc( (float**)&d_A, nBytes );
  hipMalloc( (float**)&d_B, nBytes );
  hipMalloc( (float**)&d_C, nBytes );
  
  /* hostメモリからdeviceメモリにデータ転送 */
  hipMemcpy( d_A, h_A, nBytes, hipMemcpyHostToDevice );
  hipMemcpy( d_B, h_B, nBytes, hipMemcpyHostToDevice );
  hipMemcpy( d_C, h_C, nBytes, hipMemcpyHostToDevice );

  /* スレッド数の宣言 */
  dim3 block( ARRAY_SIZE_X );
  dim3 grid( ARRAY_SIZE_Y );

   /* set the time */
  struct timeval tv, pe;
  gettimeofday(&tv, NULL);
  
  /* カーネル関数の呼び出し */
  sumArraysOnGPU<<< block,grid >>>(d_A, d_B, d_C);
    
  /* 同期処理 */
  hipDeviceSynchronize();
  
  /*stop the timer*/
  gettimeofday(&pe, NULL);
  float timer = pe.tv_sec - tv.tv_sec +(pe.tv_usec-tv.tv_usec)*1e-6;
  printf("%f sec\n",timer);
  
  /* deviceメモリからhostメモリにデータ転送 */
  hipMemcpy( h_C, d_C, nBytes, hipMemcpyDeviceToHost );

  
  
  /* 計算結果が合っているかチェック */
  checkResult(h_C, total_array_size);
  
  /* ホストメモリの解放 */ 
  free(h_A);
  free(h_B);
  free(h_C);
  
  /* デバイスメモリの解放 */
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);


  return 0;
}
